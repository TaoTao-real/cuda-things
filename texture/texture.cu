#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <algorithm>

using namespace std;

texture<float, hipTextureType2D, hipReadModeElementType> texRefA;
texture<float, hipTextureType2D, hipReadModeElementType> texRefB;

__global__ void mmul_tex(float* DC, int n){
    int col = threadIdx.x + blockIdx.x*blockDim.x;
    int row = threadIdx.y + blockIdx.y*blockDim.x;
    __shared__ float tiledAMatrix[BLOCK_WIDTH*BLOCK_WIDTH];
    __shared__ float tiledBMatrix[BLOCK_WIDTH*BLOCK_WIDTH];
    int t = (n+blockDim.x-1)/blockDim.x;
    float sum = 0;
    for(int i = 0; i < t; ++i){
        tiledAMatrix[threadIdx.x+threadIdx.y*blockDim.x] = DA[row*n + i*blockDim.x + threadIdx.x];
        tiledBMatrix[threadIdx.x+threadIdx.y*blockDim.x] = DB[(i*blockDim.y+threadIdx.y)*n+col];
        __syncthreads();
        for(int j = 0; j < blockDim.x; ++j){
            sum += tiledAMatrix[threadIdx.y*blockDim.x+j] * tiledBMatrix[blockDim.x*j+blockIdx.x];
        }
    }
    
    DC[row*n+col] = sum;
}

int main(){
    int n;
    cin >> n;
    int maxtrixSizeByByte = n*n*sizeof(float);
    float* HA = (float*) malloc(maxtrixSizeByByte);
    float* HB = (float*) malloc(maxtrixSizeByByte);
    float* HC = (float*) malloc(maxtrixSizeByByte);
    for(int i = 0; i < n; ++i){
        for(int j = 0; j < n; ++j){
            HA[i*n + j] = 1;
            HB[i*n + j] = 1;
        }
    }


    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, cudaChannelFormatFloat);
    hipArray* cuArrayA;
    hipMallocArray(&cuArrayA, &channelDesc, n, n);
    hipMemcpyToArray(cuArrayA, 0, 0, HA, maxtrixSizeByByte, hipMemcpyHostToDevice);
    hipArray* cuArrayB;
    hipMallocArray(&cuArrayB, &channelDesc, n, n);
    hipMemcpyToArray(cuArrayB, 0, 0, HB, maxtrixSizeByByte, hipMemcpyHostToDevice);

    cudaBindtexture(NULL, texRefA, cuArrayA, maxtrixSizeByByte);
    cudaBindtexture(NULL, texRefB, cuArrayB, maxtrixSizeByByte);


    float* DC;
    hipMalloc(&DC, maxtrixSizeByByte);
    
    
}