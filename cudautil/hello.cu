
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

__global__ void hello(){
	printf("hello world from GPU!\n");
	return;
}

int main(){
	hello<<<1,1>>>();
	hipError_t err = hipDeviceSynchronize();
	if(err != hipSuccess){
		printf("kernel launch failed with error %s\n", hipGetErrorString(err));
	}
	return 0;
}
